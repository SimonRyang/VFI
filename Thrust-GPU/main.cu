#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////
///
/// @file main.cu
///
/// @brief File containing main main function for the VFI problem.
///
//////////////////////////////////////////////////////////////////////////////

#include "global.h"
#include "auxFuncs.h"
#include <iostream>
#include <ctime>
#include "functors.hpp"
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/for_each.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

using namespace std;

//////////////////////////////////////////////////////////////////////////////
///
/// @fn main()
///
/// @brief Main function for the VFI problem.
///
/// @details This function solves a standard neoclassical growth model with
/// value function iteration, using Thrust. Parallelization occurs at the
/// grid of values for the state space, with each thread finding the
/// maximum of the Bellman objective function for a pair of state values.
///
/// @details See Aldrich, Eric M., Jesus Fernandez-Villaverde,
/// A. Ronald Gallant and Juan F. Rubio-Ramirez (2011), "Tapping the
/// supercomputer under your desk: Solving dynamic equilibrium models with
/// graphics processors", Journal of Economic Dynamics & Control, 35, 386-393.
///
/// @see functors.hpp
///
/// @returns 0 upon successful completion, 1 otherwise.
///
/// @author Eric M. Aldrich \n
///         ealdrich@ucsc.edu
///
/// @version 1.0
///
/// @date 12 July 2012
///
/// @copyright Copyright Eric M. Aldrich 2012 \n
///            Distributed under the Boost Software License, Version 1.0
///            (See accompanying file LICENSE_1_0.txt or copy at \n
///            http://www.boost.org/LICENSE_1_0.txt)
///
//////////////////////////////////////////////////////////////////////////////
int main()
{ 

  // Admin
  //int imax;
  REAL diff = 1.0;

  // Allocate variables in device memory
  thrust::device_vector<REAL> K(nk);
  thrust::device_vector<REAL> Z(nz);
  thrust::device_vector<REAL> P(nz*nz);
  thrust::device_vector<REAL> V(nk*nz);
  thrust::device_vector<REAL> G(nk*nz);
  thrust::device_vector<REAL> V0(nk*nz);
  thrust::device_vector<REAL>::iterator maxIter;

  // Compute TFP grid (Z)
  double lambda = 3;
  thrust::counting_iterator<int> counter(0);
  thrust::transform(counter, counter+nz,
		    Z.begin(), // output destination
		    ar1Vals<REAL>(nz, lambda, mu, sigma, rho));

  // Compute transition matrix (P)
  thrust::for_each(counter, counter+nz,
		   transMat<REAL>(nz, mu, sigma, rho,
				  raw_pointer_cast(&Z[0]), raw_pointer_cast(&P[0])));

  // Compute capital grid (K)
  thrust::transform(counter, counter+nk,
		    K.begin(), // output destination
		    kGrid<REAL>(nk, nz, beta, alpha, delta, raw_pointer_cast(&Z[0])));

  // Initialize value function
  thrust::for_each(counter, counter+nz,
  		   vfInit<REAL>(nk, eta, beta, alpha, delta,
				raw_pointer_cast(&Z[0]), raw_pointer_cast(&V0[0])));

  // iterate on the value function
  int count = 0;
  bool how = false;
  REAL tic = curr_second(); // Start counting time needed to compute the solution
  while(fabs(diff) > tol){
    if(count < 3 | count % howard == 0) how = false; else how = true;
    thrust::for_each(counter, counter+nk*nz,
		     vfStep<REAL>(nk, nz, eta, beta, alpha, delta, maxtype, how,
				  raw_pointer_cast(&K[0]), raw_pointer_cast(&Z[0]),  
				  raw_pointer_cast(&P[0]), raw_pointer_cast(&V0[0]), 
				  raw_pointer_cast(&V[0]), raw_pointer_cast(&G[0])));
    thrust::transform(V.begin(), V.end(), V0.begin(), V0.begin(), abs_diff<REAL>());
    maxIter = thrust::max_element(V0.begin(), V0.end());
    diff = *maxIter;
    V0 = V;
    ++count;
    //cout << "Iteration: " << count << ", Diff: " << diff << endl;
  }
  REAL toc = curr_second();
  cout << "Solution Time: " << toc - tic << endl;
  V = V0;

  thrust::host_vector<REAL> hV = V;
  printMatrix<REAL>(1, nk, nz, &hV[0], 4, nz, 10);
  
  return 0;

}
