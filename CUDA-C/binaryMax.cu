#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////
///
/// @file binaryMax.cpp
///
/// @brief File containing binary search maximization CUDA device function.
///
/// @author Eric M. Aldrich \n
///         ealdrich@ucsc.edu
///
/// @version 1.0
///
/// @date 23 Oct 2012
///
/// @copyright Copyright Eric M. Aldrich 2012 \n
///            Distributed under the Boost Software License, Version 1.0
///            (See accompanying file LICENSE_1_0.txt or copy at \n
///            http://www.boost.org/LICENSE_1_0.txt)
///
//////////////////////////////////////////////////////////////////////////////

#include "global.h"

//////////////////////////////////////////////////////////////////////////////
///
/// @brief CUDA device function to compute maximum of Bellman objective via
/// binary search.
///
/// @details This function finds the maximum and argmax of the Bellman
/// objective over a specified subgrid of capital by using a binary search
/// algorithm. The algorithm requires concavity.
///
/// @param [in] klo Lower index of the capital grid to begin search.
/// @param [in] nksub Number of points in the capital grid to include in
/// search.
/// @param [in] nz Length of TFP grid.
/// @param [in] ydepK value of output plus depreciated capital.
/// @param [in] eta Coefficient of relative risk aversion.
/// @param [in] beta Time discount factor.
/// @param [in] K Grid of capital values.
/// @param [in] P TFP transition matrix.
/// @param [in] V0 Current value function.
/// @param [out] V Updated value function.
/// @param [out] G Updated policy function.
///
/// @returns Void.
///
//////////////////////////////////////////////////////////////////////////////
__device__ void binaryMax(const int klo, const int nksub, const int nz,
			  const REAL ydepK, const REAL eta,
			  const REAL beta, const REAL* K, const REAL* P,
			  const REAL* V0, REAL* V, REAL* G)
{
  // binary search to find the vf max over K'
  // we assume that the value funtion is concave in capital
  int kslo, kshi, ksmid1, ksmid2, l;
  REAL Exp1, Exp2, w1, w2;
  kslo = 0;
  kshi = nksub-1;
   
  // case 1: capital grid has more than three values
  if(nksub > 3){
    // while the grid has 3 values or more, compute vf at midpoints
    // and revise the bounds of the grid
    while(kshi-kslo > 2){
      ksmid1 = (kslo + kshi)/2;
      ksmid2 = ksmid1+1;
      Exp1 = 0.0;
      Exp2 = 0.0;
      for(l = 0 ; l < nz ; ++l){
  	Exp1 += (*(P+l))*(*(V0+ksmid1*nz+l));
  	Exp2 += (*(P+l))*(*(V0+ksmid2*nz+l));
      }
      w1 = pow(ydepK-K[klo+ksmid1],1-eta)/(1-eta) + beta*Exp1;
      w2 = pow(ydepK-K[klo+ksmid2],1-eta)/(1-eta) + beta*Exp2;
      if(w2 > w1) kslo = ksmid1; else kshi = ksmid2;
    }
    // when the grid is reduced to three values, find the max
    if(w2 > w1){
      Exp1 = 0.0;
      for(l = 0 ; l < nz ; ++l) Exp1 += (*(P+l))*(*(V0+kshi*nz+l));
      w1 = pow(ydepK-K[klo+kshi],1-eta)/(1-eta) + beta*Exp1;
      if(w2 > w1){
  	*V = w2; *G = klo+kslo+1;
      } else {
  	*V = w1; *G = klo+kshi;
      }
    } else {
      Exp2 = 0.0;
      for(l = 0 ; l < nz ; ++l) Exp2 += (*(P+l))*(*(V0+kslo*nz+l));
      w2 = pow(ydepK-K[klo+kslo],1-eta)/(1-eta) + beta*Exp2;
      if(w2 > w1){
  	*V = w2; *G = klo+kslo;
      } else {
  	*V = w1; *G = klo+kslo+1;
      }
    }
  	
  // case 2: capital grid has three values
  } else if(nksub == 3) {
    // evaluate vf at each value and determine max
    Exp1 = 0.0, Exp2 = 0.0;
    REAL Exp3 = 0.0;
    for(l = 0 ; l < nz ; ++l){
      Exp1 += (*(P+l))*(*(V0+kslo*nz+l));
      Exp2 += (*(P+l))*(*(V0+kslo+1*nz+l));
      Exp3 += (*(P+l))*(*(V0+kshi*nz+l));
    }
    w1 = pow(ydepK-K[klo+kslo],1-eta)/(1-eta) + beta*Exp1;
    w2 = pow(ydepK-K[klo+kslo+1],1-eta)/(1-eta) + beta*Exp2;
    const REAL w3 = pow(ydepK-K[klo+kshi],1-eta)/(1-eta) + beta*Exp3;
    *V = w1;
    *G = klo+kslo;
    if(w2 > *V){*V = w2; *G = klo+kslo+1;}
    if(w3 > *V){ *V = w3; *G = klo+kshi;}
  	
  // case 3: capital grid has one or two values
  } else {
    Exp1 = 0.0, Exp2 = 0.0;
    for(l = 0 ; l < nz ; ++l){
      Exp1 += (*(P+l))*(*(V0+kslo*nz+l));
      Exp2 += (*(P+l))*(*(V0+kshi*nz+l));
    }
    // evaluate vf at each value and determine max
    w1 = pow(ydepK-K[klo+kslo],1-eta)/(1-eta) + beta*Exp1;
    w2 = pow(ydepK-K[klo+kshi],1-eta)/(1-eta) + beta*Exp2;
    if(w2 > w1){
      *V = w2; *G = klo+kshi;
    } else {
      *V = w1; *G = klo+kslo;
    }
  }
}
