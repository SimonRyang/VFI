#include "hip/hip_runtime.h"
#include "global.h"
#include "auxFuncs.h"
#include "hipblas.h"
#include <iostream>
#include <ctime>
#include <typeinfo>

using namespace std;

#include "ar1.cu"
#include "kGrid.cu"
#include "vfInit.cu"
#include "vfStep.cu"

//////////////////////////////////////////////////////////////////////////////
///
/// @fn main()
///
/// @brief Main function for the VFI problem.
///
/// @details This function performs value function iteration on the GPU,
/// finding the maximum of the Bellman objective function for each node in
/// the state space and iterating until convergence.
///
/// @returns 0 upon successful complete, 1 otherwise.
///
/// @author Eric M. Aldrich \n
///         ealdrich@ucsc.edu
///
/// @version 1.0
///
/// @date 24 July 2012
///
/// @copyright Copyright Eric M. Aldrich 2012 \n
///            Distributed under the Boost Software License, Version 1.0
///            (See accompanying file LICENSE_1_0.txt or copy at \n
///            http://www.boost.org/LICENSE_1_0.txt)
///
//////////////////////////////////////////////////////////////////////////////
int main()
{ 

  // admin
  int imax;
  REAL diff = 1.0;
  hipblasHandle_t handle;
  hipblasStatus_t status;
  status = hipblasCreate(&handle);
  REAL negOne = -1.0;

  // pointers to variables in device memory
  REAL* K;
  REAL* Z;
  REAL* P;
  REAL* V0;
  REAL* V;
  REAL* G;
  REAL* Vtemp;

  // allocate variables in device memory
  size_t sizeK = nk*sizeof(REAL);
  size_t sizeZ = nz*sizeof(REAL);
  size_t sizeP = nz*nz*sizeof(REAL);
  size_t sizeV = nk*nz*sizeof(REAL);
  size_t sizeG = nk*nz*sizeof(REAL);
  clock_t start = clock();
  hipMalloc((void**)&K, sizeK);
  hipMalloc((void**)&Z, sizeZ);
  hipMalloc((void**)&P, sizeP);
  hipMalloc((void**)&V0, sizeV);
  hipMalloc((void**)&Vtemp, sizeV);
  hipMalloc((void**)&V, sizeV);
  hipMalloc((void**)&G, sizeG);
  cout << "GPU Memory Allocation: " << (clock() - start)/(REAL)CLOCKS_PER_SEC << endl;

  // blocking
  dim3 dimBlockZ(nz, 1);
  dim3 dimBlockK(block_size,1);
  dim3 dimBlockV(block_size, nz);
  dim3 dimGridZ(1,1);
  dim3 dimGridK(nk/block_size,1);
  dim3 dimGridV(nk/block_size,1);

  // compute TFP grid, capital grid and initial VF
  REAL lambda = 3;
  ar1GPU<<<dimGridZ,dimBlockZ>>>(nz,lambda,mu,sigma,rho,Z,P);
  kGridGPU<<<dimGridK,dimBlockK>>>(nk,nz,beta,alpha,delta,Z,K);
  vfInitGPU<<<dimGridV,dimBlockV>>>(nz,eta,beta,alpha,delta,Z,V0);

  // iterate on the value function
  int count = 0;
  bool how = false;
  start = clock();
  while(fabs(diff) > tol){
    if(count < 3 | count % howard == 0) how = false; else how = true;
    vfStepGPU<<<dimGridV,dimBlockV>>>(nk,nz,eta,beta,alpha,delta,maxtype,how,K,Z,P,V0,V,G);
    if(typeid(realtype) == typeid(singletype)){
      status = hipblasSaxpy(handle, nk*nz, (float*)&negOne, (float*)V, 1, (float*)V0, 1);
      status = hipblasIsamax(handle, nk*nz, (float*)V0, 1, &imax);
    } else if(typeid(realtype) == typeid(doubletype)){
      status = hipblasDaxpy(handle, nk*nz, (double*)&negOne, (double*)V, 1, (double*)V0, 1);
      status = hipblasIdamax(handle, nk*nz, (double*)V0, 1, &imax);
    }
    hipMemcpy(&diff, V0+imax, sizeof(REAL), hipMemcpyDeviceToHost);
    Vtemp = V0;
    V0 = V;
    V = Vtemp;
    ++count;
  }
  cout << "GPU Solve Time: " << (clock() - start)/(REAL)CLOCKS_PER_SEC << endl;
  V = V0;
  
  // copy value and policy functions to host memory
  REAL* V = new REAL[nk*nz];
  REAL* G = new REAL[nk*nz];
  hipMemcpy(hV, V, sizeV, hipMemcpyDeviceToHost);
  hipMemcpy(hG, G, sizeG, hipMemcpyDeviceToHost);

  // copy state variable grids and transition matrix to host memory
  REAL* hK = new REAL[nk];
  REAL* hZ = new REAL[nz];
  REAL* hP = new REAL[nz*nz];
  hipMemcpy(hK, K, sizeK, hipMemcpyDeviceToHost);
  hipMemcpy(hZ, Z, sizeZ, hipMemcpyDeviceToHost);
  hipMemcpy(hP, P, sizeP, hipMemcpyDeviceToHost);

  // free variables in device memory
  hipFree(K);
  hipFree(Z);
  hipFree(P);
  hipFree(V0);
  hipFree(V);
  hipFree(Vtemp);
  hipFree(G);
  hipblasDestroy(handle);

  printMatrix<REAL>(0, nk, nz, &hV[0], 4, nz, 10);

  return 0;

}
