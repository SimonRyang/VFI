#include "hip/hip_runtime.h"
/*============================================================================

 Function      vfiGPU

 Usage         vfiGPU(hV, hG)

 Arguments     hV: pointer to array of REALs storing the value function.
                   
               hG: pointer to array of REALs storing the policy function.
	              
 Description   This function performs value function iteration on the GPU,
               finding the maximum of the Bellman objective function for each
	       node in the state space and iterating until convergence.

 Dependencies  Global Variables: eta, beta, alpha, delta, mu, rho, sigma,
                                 block_size, nk, nz, tol, maxtype, howard
				 (globalvas.h).

               Functions:        pow (math.h);
	                         cblas(S,D)axpy, cblasI(s,d)amax (cblas.h).

	       Kernels:          ar1 (ar1.cu), kGrid (kGrid.cu),
	                         vfInit(vfInit.cu), vfStep (vfStep.cu).

 Return value  Returns 0 upon successful completion, 1 otherwise.

 =============================================================================

 Author:       Eric M. Aldrich

 Contact:      ealdrich@gmail.com

 Date:         28 July 2011

 ============================================================================*/

#include "globalvars.h"
#include "hipblas.h"
#include <iostream>
#include <ctime>
#include <typeinfo>

using namespace std;

#include "ncdfgpu.cu"
#include "binary_val.cu"
#include "ar1.cu"
#include "kGrid.cu"
#include "vfInit.cu"
#include "grid_max.cu"
#include "binary_max.cu"
#include "vfStep.cu"

int vfiGPU(REAL* hV, REAL* hG) 
{ 

  int imax;
  REAL diff = 1.0;

  // pointers to variables in device memory
  REAL* K;
  REAL* Z;
  REAL* P;
  REAL* V0;
  REAL* V;
  REAL* G;
  REAL* Vtemp;

  // allocate variables in device memory
  size_t sizeK = nk*sizeof(REAL);
  size_t sizeZ = nz*sizeof(REAL);
  size_t sizeP = nz*nz*sizeof(REAL);
  size_t sizeV = nk*nz*sizeof(REAL);
  size_t sizeG = nk*nz*sizeof(REAL);
  clock_t start = clock();
  hipMalloc((void**)&K, sizeK);
  hipMalloc((void**)&Z, sizeZ);
  hipMalloc((void**)&P, sizeP);
  hipMalloc((void**)&V0, sizeV);
  hipMalloc((void**)&Vtemp, sizeV);
  hipMalloc((void**)&V, sizeV);
  hipMalloc((void**)&G, sizeG);
  cout << "GPU Memory Allocation: " << (clock() - start)/(REAL)CLOCKS_PER_SEC << endl;

  // blocking
  dim3 dimBlockZ(nz, 1);
  dim3 dimBlockK(block_size,1);
  dim3 dimBlockV(block_size, nz);
  dim3 dimGridZ(1,1);
  dim3 dimGridK(nk/block_size,1);
  dim3 dimGridV(nk/block_size,1);

  // compute TFP grid, capital grid and initial VF
  REAL lambda = 3;
  ar1<<<dimGridZ,dimBlockZ>>>(nz,lambda,mu,sigma,rho,Z,P);
  kGrid<<<dimGridK,dimBlockK>>>(nk,nz,beta,alpha,delta,Z,K);
  vfInit<<<dimGridV,dimBlockV>>>(nz,eta,beta,alpha,delta,Z,V0);

  // iterate on the value function
  int count = 0;
  bool how = false;
  start = clock();
  while(fabs(diff) > tol){
    if(count < 3 | count % howard == 0) how = false; else how = true;
    vfStep<<<dimGridV,dimBlockV>>>(nk,nz,eta,beta,alpha,delta,maxtype,how,K,Z,P,V0,V,G);
    if(typeid(realtype) == typeid(singletype)){
      hipblasSaxpy(nk*nz, -1.0, (float*)V, 1, (float*)V0, 1);
      imax = hipblasIsamax(nk*nz, (float*)V0, 1);
    } else if(typeid(realtype) == typeid(doubletype)){
      hipblasDaxpy(nk*nz, -1.0, (double*)V, 1, (double*)V0, 1);
      imax = hipblasIdamax(nk*nz, (double*)V0, 1);
    }
    hipMemcpy(&diff, V0+imax, sizeof(REAL), hipMemcpyDeviceToHost);
    Vtemp = V0;
    V0 = V;
    V = Vtemp;
    ++count;
  }
  cout << "GPU Solve Time: " << (clock() - start)/(REAL)CLOCKS_PER_SEC << endl;
  V = V0;
  
  // copy value and policy functions to host memory
  hipMemcpy(hV, V, sizeV, hipMemcpyDeviceToHost);
  hipMemcpy(hG, G, sizeG, hipMemcpyDeviceToHost);

  // copy state variable grids and transition matrix to host memory
  REAL* hK = new REAL[nk];
  REAL* hZ = new REAL[nz];
  REAL* hP = new REAL[nz*nz];
  hipMemcpy(hK, K, sizeK, hipMemcpyDeviceToHost);
  hipMemcpy(hZ, Z, sizeZ, hipMemcpyDeviceToHost);
  hipMemcpy(hP, P, sizeP, hipMemcpyDeviceToHost);

  // free variables in device memory
  hipFree(K);
  hipFree(Z);
  hipFree(P);
  hipFree(V0);
  hipFree(V);
  hipFree(Vtemp);
  hipFree(G);

  return 0;

}
