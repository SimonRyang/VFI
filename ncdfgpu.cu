#include "hip/hip_runtime.h"
/*============================================================================

 Function      ncdf

 Usage         ncdf(q)

 Arguments     q: constant REAL representing a quantile of the normal
                  density.
                   
 Description   This is a CUDA device function that computes the normal CDF at
               q.

 Return value  REAL, representing the value of the normal CDF.

 =============================================================================

 Author:       Eric M. Aldrich

 Contact:      ealdrich@gmail.com

 Date:         28 July 2011

 ============================================================================*/

// normal cdf
__device__ REAL ncdfgpu(const REAL q)
{

 // quadrature nodes
  const REAL x1 = 0.9739065285;
  const REAL x2 = -0.9739065285;
  const REAL x3 = 0.8650633666;
  const REAL x4 = -0.8650633666;
  const REAL x5 = 0.6794095682;
  const REAL x6 = -0.6794095682;
  const REAL x7 = 0.4333953941;
  const REAL x8 = -0.4333953941;
  const REAL x9 = 0.1488743389;
  const REAL x10 = -0.1488743389;

  // quadrature weights
  const REAL w1 = 0.06667134430;
  const REAL w2 = 0.06667134430;
  const REAL w3 = 0.1494513491;
  const REAL w4 = 0.1494513491;
  const REAL w5 = 0.2190863625;
  const REAL w6 = 0.2190863625;
  const REAL w7 = 0.2692667193;
  const REAL w8 = 0.2692667193;
  const REAL w9 = 0.2955242247;
  const REAL w10 = 0.2955242247;

  // symmetry
  REAL c;
  if(q < 0){
    c = -q;
  } else {
    c = q;
  }

  REAL integral;
  // bound
  if(q >= 5.2){
    integral = 1.0;
  } else if(q <= -5.2) {
    integral = 0.0;
  } else {

    // function values
    const REAL pi = 3.14159265;
    const REAL mult = 1/sqrt(2*pi);
    const REAL f1 = mult*exp(-0.5*pow(0.5*(x1+1)*c, 2));
    const REAL f2 = mult*exp(-0.5*pow(0.5*(x2+1)*c, 2));
    const REAL f3 = mult*exp(-0.5*pow(0.5*(x3+1)*c, 2));
    const REAL f4 = mult*exp(-0.5*pow(0.5*(x4+1)*c, 2));
    const REAL f5 = mult*exp(-0.5*pow(0.5*(x5+1)*c, 2));
    const REAL f6 = mult*exp(-0.5*pow(0.5*(x6+1)*c, 2));
    const REAL f7 = mult*exp(-0.5*pow(0.5*(x7+1)*c, 2));
    const REAL f8 = mult*exp(-0.5*pow(0.5*(x8+1)*c, 2));
    const REAL f9 = mult*exp(-0.5*pow(0.5*(x9+1)*c, 2));
    const REAL f10 = mult*exp(-0.5*pow(0.5*(x10+1)*c, 2));

    if(q < 0){
      integral = 0.5 - 0.5*c*(w1*f1+w2*f2+w3*f3+w4*f4+w5*f5+w6*f6+w7*f7+w8*f8+w9*f9+w10*f10);
    } else {
      integral = 0.5 + 0.5*c*(w1*f1+w2*f2+w3*f3+w4*f4+w5*f5+w6*f6+w7*f7+w8*f8+w9*f9+w10*f10);
    }

    // enforce bounds
    if(integral > 1) integral = 1;
    if(integral < 0) integral = 0;
  }

  return integral;
}
